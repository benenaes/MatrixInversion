#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Utilities.cuh"

#include <thrust/device_vector.h>

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <utility>


#include <hiprand.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void GpuGenerateRandomVector(
	float * devVector,
	const unsigned int numElements)
{
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;

	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) std::clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, devVector, numElements);
}

void GpuGenerateRandomVector(
	thrust::device_vector<float> devVector)
{
	GpuGenerateRandomVector(thrust::raw_pointer_cast(devVector.data()), devVector.size());
}

__global__ 
void GpuInitIdentity(
	float *devMatrix, 
	const unsigned int numRows, 
	const unsigned int numCols) 
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (y < numRows && x < numCols) 
	{
		if (x == y)
		{
			devMatrix[IDX2C(x, y, y)] = 1.0;
		}
		else
		{
			devMatrix[IDX2C(x, y, y)] = 0.0;
		}
	}
}

void GpuBlasMatrixMultiply(
	hipblasHandle_t &handle, 
	const float *A, 
	const float *B, 
	float *C, 
	const int m, 
	const int k, 
	const int n) 
{
	const int lda = m;
	const int ldb = k;
	const int ldc = m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

std::pair<float*, float**>
GpuCreatePositiveInvertibleSquareMatrices(
	hipblasHandle_t &handle,
	const unsigned int dimension,
	const unsigned int numMatrices)
{
	//thrust::device_vector<float> devRandomMatrices(dimension * dimension * numMatrices);
	float* devRandomMatrices;
	gpuErrchk(hipMalloc((void**)&devRandomMatrices, dimension * dimension * numMatrices * sizeof(float)));

	float* devInvertibleMatrices;
	gpuErrchk(hipMalloc((void**)&devInvertibleMatrices, dimension * dimension * numMatrices * sizeof(float)));

	//float* rawRandomMatrix = thrust::raw_pointer_cast(devRandomMatrix.data());
	//float* rawOutputMatrix = thrust::raw_pointer_cast(devOutputMatrix.data());

	//GpuGenerateRandomVector(thrust::raw_pointer_cast(devRandomMatrices.data()), dimension * dimension * numMatrices);
	GpuGenerateRandomVector(devRandomMatrices, dimension * dimension * numMatrices);

	float** devRandomMatrixPointers;
	gpuErrchk(hipMalloc((void**)&devRandomMatrixPointers, numMatrices * sizeof(float *)));

	float** devInvertibleMatrixPointers;
	gpuErrchk(hipMalloc((void**)&devInvertibleMatrixPointers, numMatrices * sizeof(float *)));

	std::vector<float*> randomMatrixPointers(numMatrices);
	std::vector<float*> invertibleMatrixPointers(numMatrices);

	for (unsigned int index = 0; index < numMatrices; ++index)
	{
		//randomMatrixPointers[index] = thrust::raw_pointer_cast(devRandomMatrices.data()) + (index * dimension * dimension);
		randomMatrixPointers[index] = devRandomMatrices + (index * dimension * dimension);
		invertibleMatrixPointers[index] = devInvertibleMatrices + (index * dimension * dimension);
	}

	gpuErrchk(hipMemcpy(devRandomMatrixPointers, randomMatrixPointers.data(), numMatrices * sizeof(float *), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(devInvertibleMatrixPointers, invertibleMatrixPointers.data(), numMatrices * sizeof(float *), hipMemcpyHostToDevice));

	const float alpha = 1;
	const float beta = 0;

	// transpose(random matrix) * random matrix leads into an invertible, positive definite matrix
	cublasSafeCall(
		hipblasSgemmBatched(
			handle, 
			HIPBLAS_OP_T, 
			HIPBLAS_OP_N, 
			dimension, 
			dimension, 
			dimension, 
			&alpha, 
			(const float**)devRandomMatrixPointers,
			dimension, 
			(const float**)devRandomMatrixPointers,
			dimension, 
			&beta, 
			devInvertibleMatrixPointers,
			dimension,
			numMatrices));

	gpuErrchk(hipFree(devRandomMatrices));
	gpuErrchk(hipFree(devRandomMatrixPointers));

	return std::make_pair(devInvertibleMatrices, devInvertibleMatrixPointers);
}

int main()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << "\n";
	std::cout << "Max global memory: " << prop.totalGlobalMem << "\n";

	// Rows = cols
	const unsigned int matrixDimension = 64;

	const unsigned int bytesPerMatrix = matrixDimension * matrixDimension * sizeof(float);
	// Input matrix, output matrix
	const unsigned int matricesNeededPerCalculation = 2;
	// Max calculations in one batch
	const float maxMatrixCalculations = static_cast<float>(prop.totalGlobalMem) / (matricesNeededPerCalculation * bytesPerMatrix);
	std::cout << "Max calculations in one batch: " << maxMatrixCalculations << "\n";

	const unsigned int numMatrices = 20000;

	hipblasHandle_t cuBlasHandle = nullptr;
	cublasSafeCall(hipblasCreate(&cuBlasHandle));

	std::pair<float*, float**> devRandomInvertibleMatricesDesc = GpuCreatePositiveInvertibleSquareMatrices(cuBlasHandle, matrixDimension, numMatrices);

	int* devPivotArray; 
	gpuErrchk(hipMalloc((void**)&devPivotArray, matrixDimension * numMatrices * sizeof(int)));

	int* devInfoArray;  
	gpuErrchk(hipMalloc((void**)&devInfoArray, numMatrices * sizeof(int)));

	cublasSafeCall(
		hipblasSgetrfBatched(
			cuBlasHandle, 
			matrixDimension, 
			devRandomInvertibleMatricesDesc.second,
			matrixDimension, 
			devPivotArray, 
			devInfoArray, 
			numMatrices));

	int* infoArray = (int*) malloc(numMatrices * sizeof(int));

	gpuErrchk(hipMemcpy(infoArray, devInfoArray, numMatrices * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < numMatrices; ++i)
	{
		if (infoArray[i] != 0) 
		{
			std::cerr << "Factorization of matrix " << i << " failed: Matrix may be singular\n";
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}
	}

	// --- Allocate host space for the inverted matrices 
	float *invertedMatrices = new float[matrixDimension * matrixDimension * numMatrices];

	// --- Allocate device space for the inverted matrices 
	float *devInvertedMatrices; 
	gpuErrchk(hipMalloc((void**)&devInvertedMatrices, matrixDimension * matrixDimension * numMatrices * sizeof(float)));

	// --- Creating the array of pointers needed as output to the batched getri
	float **invertedMatrixPointers = (float **)malloc(numMatrices * sizeof(float *));
	for (int i = 0; i < numMatrices; ++i)
	{
		invertedMatrixPointers[i] = (float *)((char*)devInvertedMatrices + i*((size_t)matrixDimension*matrixDimension) * sizeof(float));
	}

	float **devInvertedMatrixPointers;
	gpuErrchk(hipMalloc((void**)&devInvertedMatrixPointers, numMatrices * sizeof(float *)));
	gpuErrchk(hipMemcpy(devInvertedMatrixPointers, invertedMatrixPointers, numMatrices * sizeof(float *), hipMemcpyHostToDevice));
	free(invertedMatrixPointers);

	cublasSafeCall(
		hipblasSgetriBatched(
			cuBlasHandle, 
			matrixDimension, 
			(const float **)devRandomInvertibleMatricesDesc.second,
			matrixDimension,
			devPivotArray,
			devInvertedMatrixPointers, 
			matrixDimension,
			devInfoArray,
			numMatrices));

	gpuErrchk(hipMemcpy(infoArray, devInfoArray, numMatrices * sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < numMatrices; ++i)
	{
		if (infoArray[i] != 0) 
		{
			std::cerr << "Inversion of matrix " << i << " failed: Matrix may be singular\n";
			hipDeviceReset();
			exit(EXIT_FAILURE);
		}
	}

	gpuErrchk(
		hipMemcpy(
			invertedMatrices, 
			devInvertedMatrices, 
			matrixDimension * matrixDimension * sizeof(float), 
			hipMemcpyDeviceToHost));

	free(infoArray);
	free(invertedMatrices);
	gpuErrchk(hipFree(devInfoArray));
	gpuErrchk(hipFree(devPivotArray));
	gpuErrchk(hipFree(devRandomInvertibleMatricesDesc.first));
	gpuErrchk(hipFree(devRandomInvertibleMatricesDesc.second));
	gpuErrchk(hipFree(devInvertedMatrices));
	gpuErrchk(hipFree(devInvertedMatrixPointers));


	//const unsigned int blockDimX = 32;
	//const unsigned int blockDimY = 32;

	//dim3 blockDim(blockDimX, blockDimY);
	//dim3 gridDim((matrixDimension + blockDimX - 1) / blockDimX, (matrixDimension + blockDimY - 1) / blockDimY);
	//GpuInitIdentity <<<gridDim, blockDim >>>(thrust::raw_pointer_cast(&devI[0]), numberRowsI, numberColsI);

	//hipsolverHandle_t cuSolverHandle = nullptr;
	//hipsolverStatus_t cuSolverStatus = hipsolverDnCreate(&cuSolverHandle);

	//cuSolverStatus = hipsolverDnDestroy(cuSolverHandle);
	cublasSafeCall(hipblasDestroy(cuBlasHandle));

    return 0;
}

